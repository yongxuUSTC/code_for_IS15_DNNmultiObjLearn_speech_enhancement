#include <stdio.h>
#include <stdlib.h>

#include <sys/time.h>
#include "BP_GPU.h"
#include "DevFunc.h"

#define THREUSEMULTIGPU 256

BP_GPU::BP_GPU(int a_GPU_selected, int a_numlayers, int *a_layersizes, int a_bunchsize, float a_lrate, float a_momentum,  
	float a_weightcost,float **weights, float **bias,int a_dropoutflag,float a_visible_omit,float a_hid_omit)
	:GPU_selected(a_GPU_selected),numlayers(a_numlayers),bunchsize(a_bunchsize),momentum(a_momentum),lrate(a_lrate),weightcost(a_weightcost),dropoutflag(a_dropoutflag), visible_omit(a_visible_omit),hid_omit(a_hid_omit)
{
	int i,j;
   int maxlayersize=0;
	//// set GPU num
	hipGetDeviceCount(&GPU_total);
	printf("Total GPU Device : %d\n",GPU_total);

	if(GPU_selected > GPU_total || GPU_selected < 1)
	{
		printf("GPU Num %d Not In Range %d-%d\n",GPU_selected,1,GPU_total);
		exit(0);
	}
	printf("Use GPU Device : %d\n",GPU_selected);



	int bunch_part[GPU_selected];
	int part = bunchsize/GPU_selected;

	for(i= 0; i< GPU_selected-1;i++)
	{
		bunch_part[i] = part;
	}
	bunch_part[GPU_selected -1] = bunchsize -part*(GPU_selected -1);



	////Init cublas && streams
	dev = (BP_WorkSpace*) malloc(GPU_selected * sizeof(BP_WorkSpace));
	handles = (hipblasHandle_t*) malloc(GPU_selected * sizeof(hipblasHandle_t)); 	
	streams = (hipStream_t*) malloc(GPU_selected * sizeof(hipStream_t));
  gen = (hiprandGenerator_t*) malloc(GPU_selected * sizeof(hiprandGenerator_t));

	for(i = 0;i < GPU_selected;i++)
	{
		hipError_t er;
    hiprandStatus_t eg;
    
		er = hipSetDevice(i);
		//er = hipSetDevice(1);
		if (er!=hipSuccess)
			printf("hipSetDevice(%d) failed\n",i);


		er =hipStreamCreate(&(streams[i]));
		if (er!=hipSuccess)
			printf("hipStreamCreate(%d) failed\n",i);

		hipblasStatus_t eb = hipblasCreate(&handles[i]);
		if (eb!=HIPBLAS_STATUS_SUCCESS)
			printf("hipblasCreate(%d) failed\n",i);

		eb = hipblasSetStream(handles[i],streams[i]);
		if (eb!=HIPBLAS_STATUS_SUCCESS)
			printf("hipblasSetStream(handles[%d],streams[%d]) failed\n",i,i);
			
	  eg = hiprandCreateGenerator(&gen[i] ,HIPRAND_RNG_PSEUDO_DEFAULT);
		if(eg!=HIPRAND_STATUS_SUCCESS)
            printf("hiprandCreateGenerator(%d) failed\n",i);

		eg = hiprandSetStream(gen[i],streams[i]);
		if(eg!=HIPRAND_STATUS_SUCCESS)
            printf("hiprandSetStream(%d) failed\n",i);

		srand(unsigned(time(NULL)));
		hiprandSetPseudoRandomGeneratorSeed(gen[i] ,rand());
	}
	if(GPU_selected >1)
	{


		for(i =0;i< GPU_selected;i++)
		{

			hipSetDevice(i);
		  // hipSetDevice(1);
			for(j =0;j< GPU_selected;j++)
			{			 
		        
				if(j != i)
				{
				    int UVA;
				    hipDeviceCanAccessPeer(&UVA,j,i);
					if(UVA==0)
					{
					    printf("hipDeviceCanAccessPeer error\n");
						exit(0);
					}
					else
					{
					    printf("hipDeviceCanAccessPeer between Device %d and Device %d OK\n",j,i);
					    hipDeviceEnablePeerAccess(j, 0);
					}
					    
				}
					
			}
		}
	}

	//// Alloc device Memory
	for(i =0; i < numlayers;i++)
	{
		layersizes[i] = a_layersizes[i];
		if (maxlayersize<layersizes[i])
			{maxlayersize=layersizes[i];}
	}

	for(j =0;j< GPU_selected;j++)
	{
	    if(j==0)
		{
		    hipSetDevice(0);
		   // hipSetDevice(1);
		    devnew_vf("in", 						MAXCACHEFRAME *layersizes[0], &(dev[j].in));
		    devnew_vf("out", 						bunchsize *layersizes[numlayers -1], &(dev[j].out));
		    //devnew_vi("targ", 					MAXCACHEFRAME, &(dev[j].targ));
			devnew_vf("targ", 					MAXCACHEFRAME*layersizes[numlayers -1], &(dev[j].targ));/////////////////////////////////yongxu
         devnew_vf("DevRandVector",			maxlayersize *bunch_part[j],&(dev[j].DevRandVector));
		devnew_vi("DevSeed",BASICSIZE,&(dev[j].DevSeed));
		    for (i = 1; i< numlayers; i++)
		    {
			    devnew_vf("bias", 	 layersizes[i], &(dev[j].bias[i]));
			    devnew_vf("weights", layersizes[i] *layersizes[i-1], &(dev[j].weights[i]));
			    devnew_vf("delta_bias", 	 layersizes[i], &(dev[j].delta_bias[i]));
			    devnew_vf("delta_weights", layersizes[i] *layersizes[i-1], &(dev[j].delta_weights[i]));
			    devnew_vf("layer_y", bunchsize *layersizes[i], &(dev[j].layer_y[i]));
			    devnew_vf("layer_x", bunchsize *layersizes[i], &(dev[j].layer_x[i]));
			    devnew_vf("layer_dedy", bunchsize *layersizes[i], &(dev[j].layer_dedy[i]));
			    devnew_vf("layer_dydx", bunchsize *layersizes[i], &(dev[j].layer_dydx[i]));
			    devnew_vf("layer_dedx", bunchsize *layersizes[i], &(dev[j].layer_dedx[i]));
			    devnew_vf("layer_ydedx", layersizes[i] *layersizes[i-1], &(dev[j].layer_ydedx[i]));
			    devnew_vf("layer_sumdedx", layersizes[i], &(dev[j].layer_sumdedx[i]));
		    }		    
		}
		else
		{
		    hipSetDevice(j);
		   // hipSetDevice(1);
		    devnew_vf("in", 						MAXCACHEFRAME *layersizes[0], &(dev[j].in));
		    devnew_vf("out", 						bunch_part[j] *layersizes[numlayers -1], &(dev[j].out));
		    //devnew_vi("targ", 					MAXCACHEFRAME, &(dev[j].targ));
			devnew_vf("targ", 					MAXCACHEFRAME*layersizes[numlayers -1], &(dev[j].targ));////////////////////////////////////yongxu

		    for (i = 1; i< numlayers; i++)
		    {
			    devnew_vf("bias", 	 layersizes[i], &(dev[j].bias[i]));
			    devnew_vf("weights", layersizes[i] *layersizes[i-1], &(dev[j].weights[i]));
			    devnew_vf("delta_bias", 	 layersizes[i], &(dev[j].delta_bias[i]));
			    devnew_vf("delta_weights", layersizes[i] *layersizes[i-1], &(dev[j].delta_weights[i]));
			    devnew_vf("layer_y", bunch_part[j] *layersizes[i], &(dev[j].layer_y[i]));
			    devnew_vf("layer_x", bunch_part[j] *layersizes[i], &(dev[j].layer_x[i]));
			    devnew_vf("layer_dedy", bunch_part[j] *layersizes[i], &(dev[j].layer_dedy[i]));
			    devnew_vf("layer_dydx", bunch_part[j] *layersizes[i], &(dev[j].layer_dydx[i]));
			    devnew_vf("layer_dedx", bunch_part[j] *layersizes[i], &(dev[j].layer_dedx[i]));
			    devnew_vf("layer_ydedx", layersizes[i] *layersizes[i-1], &(dev[j].layer_ydedx[i]));
			    devnew_vf("layer_sumdedx", layersizes[i], &(dev[j].layer_sumdedx[i]));
		    }
		}

	}
	if(GPU_selected >1)
	{
		hipDeviceSynchronize();
	}

	////copy weights && biases to devices
	for(j =0;j< GPU_selected;j++)
	{

		hipSetDevice(j);
   //hipSetDevice(1);
    
		for(i = 1; i< numlayers; i++)
		{
			todev_vf_vf("weights", layersizes[i-1] *layersizes[i], weights[i], dev[j].weights[i], streams[j]);
			todev_vf_vf("bias", layersizes[i], bias[i], dev[j].bias[i], streams[j]);
		}
	}
	if(GPU_selected >1)
	{
		hipDeviceSynchronize();
	}
	printf("Created net with %d layers, bunchsize %d.\n", numlayers, bunchsize);
}

BP_GPU::~BP_GPU()
{
	int i,j;

	////streams & cublas free	
	for(j =0;j< GPU_selected;j++)
	{

		hipSetDevice(j);
    //hipSetDevice(1);

		devfree_vf("in", dev[j].in);
		devfree_vf("out", dev[j].out);
		//devfree_vi("targ", dev[j].targ);
		devfree_vf("targ", dev[j].targ);/////////////////////////////////////////////////yongxu
       devfree_vf("DevRandVector",dev[j].DevRandVector);
	 devfree_vi("DevSeed", dev[j].DevSeed);
		for (i = 1; i< numlayers; i++)
		{
			devfree_vf("weights", dev[j].weights[i]);
			devfree_vf("bias", dev[j].bias[i]);
			devfree_vf("delta_weights", dev[j].delta_weights[i]);
			devfree_vf("delta_bias", dev[j].delta_bias[i]);
			devfree_vf("layer_x", dev[j].layer_x[i]);
			devfree_vf("layer_y", dev[j].layer_y[i]);
			devfree_vf("layer_dedx", dev[j].layer_dedx[i]);
			devfree_vf("layer_dydx", dev[j].layer_dydx[i]);
			devfree_vf("layer_dedy", dev[j].layer_dedy[i]);
			devfree_vf("layer_ydedx", dev[j].layer_ydedx[i]);
			devfree_vf("layer_sumdedx", dev[j].layer_sumdedx[i]);
		}


		hipblasDestroy(handles[j]);
		hipStreamDestroy(streams[j]);
	 hiprandDestroyGenerator(gen[j]);
	}
	delete[] dev;
}

//void BP_GPU::train(int n_frames, const float* in, const int *targ)
//void BP_GPU::train(int n_frames, const float* in, const float *targ)////////////////////////////by yongxu
void BP_GPU::train(int n_frames, float* in, const float *targ)
{

	int i,j;
	//int t;
	int frames_this_bunch;	// Number of frames to handle this bunch
	int n_input = layersizes[0];
	int out_dims= layersizes[numlayers-1];
	
	float **realin = new float*[GPU_selected];
	//int **realtarg = new int*[GPU_selected];
	float **realtarg = new float*[GPU_selected];///////////////////////////////////by yongxu
	//float *realin;
	//int *realtarg;

	int n_frames_part = n_frames/GPU_selected;

// for (t=0;t<517;t++)
// { printf("in[%d]=%f,",t,in[t]);
//   }
//   printf ("\n");
//   
//   for (t=0;t<200;t++)
//   {printf("targ[%d]=%f,",t,targ[t]);
//   	}
//   printf ("\n");
   
	// First copy data to GPU
	for(i= 0; i< GPU_selected;i++)
	{

		hipSetDevice(i);
		//hipSetDevice(1);
		todev_vf_vf("in",n_frames_part * n_input, in + i* n_frames_part* n_input, dev[i].in, streams[i]);
		//todev_vi_vi("targ", n_frames_part, targ + i* n_frames_part, dev[i].targ, streams[i]);
	    todev_vf_vf("targ", n_frames_part * out_dims, targ + i* n_frames_part, dev[i].targ, streams[i]);
	}
	if(GPU_selected >1)
	{
		hipDeviceSynchronize();
	}
	//printf("Copy Data Sucess , %d Frames\n",n_frames);


	for(i= 0; i< GPU_selected;i++)
	{
		realin[i] = dev[i].in;
		realtarg[i] = dev[i].targ;    
	}

  

	//printf("GPU_selected : %d\n",GPU_selected);
	for (i=0; i< n_frames; i+= bunchsize)
	{
		//printf("i=%d\n",i);
		frames_this_bunch = (bunchsize > n_frames - i)?(n_frames - i):bunchsize;
		if(frames_this_bunch == bunchsize)
		{
			//printf("in \n");
			if(GPU_selected == 1)
				{
					//printf("in-in \n");
					//printf("realin[0][1]=%f,realtarg[0][1]=%f\n",realin[0][1],realtarg[0][1]);//����ط��䲻������Ҳ������
					//printf("dev[0].in[1]=%f,dev[0].targ[1]=%f\n",in[1],targ[1]);
					//printf("begin to run train_bunch_single\n");
					
					train_bunch_single(frames_this_bunch, realin[0], realtarg[0]);//[0]��ʾ��0��cuda device,//realin[0], realtarg[0]
			    //������ÿ��batch��ȥ�ܣ���realin��realtarg����ÿ��ָ��GPU���ÿ��batch
			    //printf("complete train_bunch_single\n");
			    }
			//else
				//train_bunch_multi(frames_this_bunch, realin, realtarg);
		}
		else
		{
			printf("this bunch has only %d samples and is ignored.\n",frames_this_bunch);
		}
		
		for(j= 0; j< GPU_selected;j++)	
		{	
			realin[j] += n_input * frames_this_bunch/GPU_selected;
			realtarg[j] += out_dims * frames_this_bunch/GPU_selected;
		}

	}
	//printf("end here before\n");
	delete[] realin;
	delete[] realtarg;
//printf("end here\n");
}

////void BP_GPU::train(int n_frames, const float* in, const int *targ)
////����д��������Ķ��GPUȥ�ܵĳ���ע�͵������ⷢ������
//void BP_GPU::train(int n_frames, const float* in, const float *targ)////////////////////////////by yongxu
//{
//
//	int i,t;
//	int frames_this_bunch;	// Number of frames to handle this bunch
//	int n_input = layersizes[0];
//	float *realin = new float[GPU_selected];
//	//int **realtarg = new int*[GPU_selected];
//	float *realtarg = new float[GPU_selected];///////////////////////////////////by yongxu
//	//float *realin;
//	//int *realtarg;
//
//	int n_frames_part = n_frames/1;
//
//// for (t=0;t<560;t++)//����check�ˣ�ƴ֡��ѵ������֡��Ӧtargetһ֡������
//// { printf("in[%d]=%f,",t,in[t]);
////   }
////   printf ("\n");
////   
////   for (t=0;t<200;t++)
////   {printf("targ[%d]=%f,",t,targ[t]);
////   	}
////   printf ("\n");
//   
//	// First copy data to GPU
//		hipSetDevice(0);
//		todev_vf_vf("in",n_frames_part * n_input, in + 0* n_frames_part* n_input, dev[0].in, streams[0]);
//		//todev_vi_vi("targ", n_frames_part, targ + i* n_frames_part, dev[i].targ, streams[i]);
//	    todev_vf_vf("targ", n_frames_part * out_dims, targ + 0* n_frames_part * out_dims, dev[0].targ, streams[0]);
//
//	printf("Copy Data Sucess , %d Frames\n",n_frames);
//
//		realin = dev[0].in;
//		realtarg = dev[0].targ;  
//
//	printf("GPU_selected : %d\n",GPU_selected);
//	for (i=0; i< n_frames; i+= bunchsize)
//	{
//		printf("i=%d\n",i);
//		frames_this_bunch = (bunchsize > n_frames - i)?(n_frames - i):bunchsize;
//		if(frames_this_bunch == bunchsize)
//		{
//			printf("in \n");
//
//					//printf("realin[0]=%f,realtarg[0]=%f\n",realin[0],realtarg[0]);//����ط��䲻������Ҳ������
//					//printf("dev[0].in[1]=%f,dev[0].targ[1]=%f\n",in[1],targ[1]);
//					printf("begin to run train_bunch_single\n");
//					
//					train_bunch_single(frames_this_bunch, realin, realtarg);//[0]��ʾ��0��cuda device
//			    printf("complete train_bunch_single\n");
//			    
//
//		}
//		else
//		{
//			printf("this bunch has only %d samples and is ignored.\n",frames_this_bunch);
//		}
//		
//
//			realin += n_input * frames_this_bunch/1;
//			realtarg += out_dims * frames_this_bunch/1;
//		
//
//	}
//	printf("this train end\n");
//	delete[] realin;
//	delete[] realtarg;
//	
//	printf("this train end 2 \n");
//
//}

//int BP_GPU::CrossValid(int n_frames, const float* in, const int *targ)
float BP_GPU::CrossValid(int n_frames, const float* in, const float *targ)/////////////////////////////////////by yongxu
{
	//only use one GPU
	//int correct_samples =0;
	float squared_err=0.0f;/////////////////////////////////////////////by yongxu
	//float squared_err_speech=0.0f;/////////////////////////////////////////////by yongxu,�������
	//float squared_err_noise=0.0f;/////////////////////////////////////////////by yongxu���������
	//int *out = new int [bunchsize];
	int out_dims= layersizes[numlayers-1];
	
  float *out = new float [bunchsize*out_dims];///////////////////////////////by yongxu, ����ط���һ����ά������batch*feadim��
	//int *out;
	//hipHostMalloc((void**)&out, bunchsize * sizeof(int));
	int i,j,d;
	//int t;
	int frames_this_bunch;	// Number of frames to handle this bunch
	int n_input = layersizes[0];//���������ά������������չ֡�ģ�
	float *realin;


//
// for (t=0;t<560;t++)//����check�ˣ�ƴ֡��ѵ������֡��Ӧtargetһ֡������
// { printf("in[%d]=%f,",t,in[t]);
//   }
//   printf ("\n");
//   
//   for (t=0;t<200;t++)
//   {printf("targ[%d]=%f,",t,targ[t]);
//   	}
//   printf ("\n");


	// First copy data to GPU
	hipSetDevice(0);
	//hipSetDevice(1);
	todev_vf_vf("in", n_frames* n_input, in, dev[0].in, streams[0]);

	realin = dev[0].in;

	FILE *fp=fopen("CV_out.txt","w");

	for (i=0; i< n_frames; i+= bunchsize)//n_frames�Ǹ�CV������֡����bunchsizeָ����һ��bunch���ж���֡��Ȼ��ÿ��bunch�ֱ����
	{
		
		frames_this_bunch = (bunchsize > n_frames - i)?(n_frames - i):bunchsize;

		//cv_bunch_single(frames_this_bunch, realin, out[i]);
		cv_bunch_single(frames_this_bunch, realin, out);

		//// compute correct_samples
		////////////compute squared error
		//fprintf(fp,"%d\n\n",frames_this_bunch);
		for(j =0; j< frames_this_bunch;j++)
		{
		  for(d=350;d<out_dims;d++)///////////////////////////////////////by yongxu, ���ǵ��������ʼ����out_dimsά
		  {
		    squared_err = squared_err + (out[j*out_dims+d]-targ[j*out_dims+d])*(out[j*out_dims+d]-targ[j*out_dims+d]);/////////////by yongxu, �ر�ע�⣺squared error����correct_samples�෴�ĸ���
			
			//fprintf(fp,"%f ",out[j*out_dims+d]);
			}
			
//		  //�����������ֵ�error
//					  for(d=0;d<out_dims/2;d++)///////////////////////////////////////by yongxu, ���ǵ��������ʼ����out_dimsά
//		  {
//		    squared_err_speech = squared_err_speech + (out[j*out_dims+d]-targ[j*out_dims+d])*(out[j*out_dims+d]-targ[j*out_dims+d]);/////////////by yongxu, �ر�ע�⣺squared error����correct_samples�෴�ĸ���
//			}
//			
//					  //�����������ֵ�error
//					  for(d=out_dims/2;d<out_dims;d++)///////////////////////////////////////by yongxu, ���ǵ��������ʼ����out_dimsά
//		  {
//		    squared_err_noise = squared_err_noise + (out[j*out_dims+d]-targ[j*out_dims+d])*(out[j*out_dims+d]-targ[j*out_dims+d]);/////////////by yongxu, �ر�ע�⣺squared error����correct_samples�෴�ĸ���
//			}
			
			//fprintf(fp,"\n");
		}

		realin += n_input * frames_this_bunch;
		targ += out_dims * frames_this_bunch;
	}

	fclose(fp);

	delete []out;
	//hipHostFree(out);
	//return correct_samples;
	return squared_err;
}

//int BP_GPU::CrossValid(int n_frames, const float* in, const int *targ)
float BP_GPU::CrossValid2(int n_frames, const float* in, const float *targ)/////////////////////////////////////by yongxu
{
	//only use one GPU
	//int correct_samples =0;
	//float squared_err=0.0f;/////////////////////////////////////////////by yongxu
	float squared_err_speech=0.0f;/////////////////////////////////////////////by yongxu,�������
	//float squared_err_noise=0.0f;/////////////////////////////////////////////by yongxu���������
	//int *out = new int [bunchsize];
	int out_dims= layersizes[numlayers-1];
	
  float *out = new float [bunchsize*out_dims];///////////////////////////////by yongxu, ����ط���һ����ά������batch*feadim��
	//int *out;
	//hipHostMalloc((void**)&out, bunchsize * sizeof(int));
	int i,j,d;
	//int t;
	int frames_this_bunch;	// Number of frames to handle this bunch
	int n_input = layersizes[0];//���������ά������������չ֡�ģ�
	float *realin;


//
// for (t=0;t<560;t++)//����check�ˣ�ƴ֡��ѵ������֡��Ӧtargetһ֡������
// { printf("in[%d]=%f,",t,in[t]);
//   }
//   printf ("\n");
//   
//   for (t=0;t<200;t++)
//   {printf("targ[%d]=%f,",t,targ[t]);
//   	}
//   printf ("\n");


	// First copy data to GPU
	hipSetDevice(0);
	//hipSetDevice(1);
	todev_vf_vf("in", n_frames* n_input, in, dev[0].in, streams[0]);

	realin = dev[0].in;

	FILE *fp=fopen("CV_out.txt","w");

	for (i=0; i< n_frames; i+= bunchsize)//n_frames�Ǹ�CV������֡����bunchsizeָ����һ��bunch���ж���֡��Ȼ��ÿ��bunch�ֱ����
	{
		
		frames_this_bunch = (bunchsize > n_frames - i)?(n_frames - i):bunchsize;

		//cv_bunch_single(frames_this_bunch, realin, out[i]);
		cv_bunch_single(frames_this_bunch, realin, out);

		//// compute correct_samples
		////////////compute squared error
		//fprintf(fp,"%d\n\n",frames_this_bunch);
		for(j =0; j< frames_this_bunch;j++)
		{
//		  for(d=0;d<out_dims;d++)///////////////////////////////////////by yongxu, ���ǵ��������ʼ����out_dimsά
//		  {
//		    squared_err = squared_err + (out[j*out_dims+d]-targ[j*out_dims+d])*(out[j*out_dims+d]-targ[j*out_dims+d]);/////////////by yongxu, �ر�ע�⣺squared error����correct_samples�෴�ĸ���
//			
//			//fprintf(fp,"%f ",out[j*out_dims+d]);
//			}
			
		  //�����������ֵ�error
					  for(d=0;d<257;d++)///////////////////////////////////////by yongxu, ���ǵ��������ʼ����out_dimsά
		  {
		    squared_err_speech = squared_err_speech + (out[j*out_dims+d]-targ[j*out_dims+d])*(out[j*out_dims+d]-targ[j*out_dims+d]);/////////////by yongxu, �ر�ע�⣺squared error����correct_samples�෴�ĸ���
			}
			
//					  //�����������ֵ�error
//					  for(d=out_dims/2;d<out_dims;d++)///////////////////////////////////////by yongxu, ���ǵ��������ʼ����out_dimsά
//		  {
//		    squared_err_noise = squared_err_noise + (out[j*out_dims+d]-targ[j*out_dims+d])*(out[j*out_dims+d]-targ[j*out_dims+d]);/////////////by yongxu, �ر�ע�⣺squared error����correct_samples�෴�ĸ���
//			}
			
			//fprintf(fp,"\n");
		}

		realin += n_input * frames_this_bunch;
		targ += out_dims * frames_this_bunch;
	}

	fclose(fp);

	delete []out;
	//hipHostFree(out);
	//return correct_samples;
	return squared_err_speech;
}


//void BP_GPU::train_bunch_single(int n_frames, const float *in, const int* targ)
//void BP_GPU::train_bunch_single(int n_frames, const float *in, const float* targ)/////////////////////by yongxu
void BP_GPU::train_bunch_single(int n_frames, float *in, const float* targ)
{
	const float one  = 1.0f;
	const float zero = 0.0f;
	//int i,j;
	int cur_layer;			// The index of the current layer.
	int prev_layer;			// The index of the previous layer.
	int cur_layer_units;	// The number of units in the current layer.
	int prev_layer_units;	// The number of units in the previous layer.
	int cur_layer_size;		// The size of the current layer.
  int prev_layer_size;
  
	float* cur_layer_x;
	float* cur_layer_y;				// Output from the current layer
	const float* prev_layer_y;	// Output from the previous non-linearity.
	float* cur_layer_dydx;	// dydx for the current layer.
	float* cur_layer_dedy;	// dedy for the current layer.
	float* prev_layer_dedy;	// dedy for the previous layer.
	float* cur_layer_dedx;	// dedx for the current layer.
	float* cur_layer_ydedx;
	float* cur_layer_sumdedx;
	float* cur_layer_bias;	// Biases for the current layer.
	float* cur_layer_delta_bias; // Delta biases for the current layer.
	float* cur_layer_delta_weights;
	float* cur_weights;		// Weights inputing to the current layer.
	float cur_lrate =  lrate;
	
	//float *out_check = new float [n_frames*out_dims];//Ϊ��check��������

 // printf("in train_bunch_single\n");
  //FILE *fp=fopen("log_train_bunch_single.txt","w");//���������д���������ѵ�����Ϊ��cuda�����Ҫ����cpu����У�


	//// Forward
	for (cur_layer=1; cur_layer< numlayers; cur_layer++)
	{
		//printf("forward ing\n");
		prev_layer = cur_layer - 1;
		cur_layer_units = layersizes[cur_layer];
		prev_layer_units = layersizes[prev_layer];
		cur_layer_size = cur_layer_units * n_frames;//batch���֡��
		prev_layer_size = prev_layer_units * n_frames;
		cur_layer_x = dev[0].layer_x[cur_layer];
		cur_layer_y = dev[0].layer_y[cur_layer];
		
		//if (cur_layer==1)//Ϊ�����dropout����ע�͵���
		//	prev_layer_y = in;
		//else
		//	prev_layer_y = dev[0].layer_y[prev_layer];
		
				if (cur_layer==1)
		{   
			if(dropoutflag==1)
			{
			  hiprandGenerateUniform(gen[0], dev[0].DevRandVector, prev_layer_size);
			  DevDropout(streams[0],prev_layer_size,visible_omit,in,dev[0].DevRandVector);
			}
			prev_layer_y = in;
		}
		else
		{
			if(dropoutflag==1)
			{
			 hiprandGenerateUniform(gen[0], dev[0].DevRandVector, prev_layer_size);
			 DevDropout(streams[0],prev_layer_size, hid_omit, dev[0].layer_y[prev_layer], dev[0].DevRandVector);
			}
			prev_layer_y = dev[0].layer_y[prev_layer];
        }
	    hipDeviceSynchronize();
		
		cur_layer_bias = dev[0].bias[cur_layer];
		cur_weights = dev[0].weights[cur_layer];

		DevMultiCopy(streams[0],n_frames, cur_layer_units, cur_layer_bias, cur_layer_x);
		SgemmNN(handles[0],cur_layer_units, prev_layer_units, n_frames, cur_weights, prev_layer_y, cur_layer_x, one, one); 

		if (cur_layer != numlayers - 1){
			DevSigmoid(streams[0],cur_layer_size, cur_layer_x, cur_layer_y);
		}
		else{  /////////////////////////////ֱ��ע�͵�������ؾ���linear�ģ�����
			//DevSoftmax(streams[0],n_frames, cur_layer_units, cur_layer_x, dev[0].out);
			//DevLinearOutCopy(streams[0],n_frames, cur_layer_units, cur_layer_x, dev[0].out);
	    //out=cur_layer_x;
	    //hipSetDevice(0);
	    //printf("come here\n");
	    hipMemcpy(dev[0].out,cur_layer_x,n_frames*cur_layer_units*sizeof(float),hipMemcpyDeviceToDevice);
	    //hipMemcpy(out_check,cur_layer_x,n_frames*cur_layer_units*sizeof(float),hipMemcpyDeviceToHost);
		
		//����������
//		for(i =0; i< n_frames;i++)
//		{
//		  for(j=0;j<out_dims;j++)///////////////////////////////////////by yongxu, ���ǵ��������ʼ����out_dimsά
//		  {
//			
//			printf("%f ",out_check[i*out_dims+j]);
//			}
//			printf("\n");exit(0);
//		}
		  //delete []out;
		}
	}

	// Backward
	for (cur_layer = numlayers -1; cur_layer >0; cur_layer--)
	{
		//printf("Backward ing\n");
		prev_layer = cur_layer - 1;
		cur_layer_units = layersizes[cur_layer];
		prev_layer_units = layersizes[prev_layer];
		cur_layer_size = cur_layer_units * n_frames;
		cur_layer_y = dev[0].layer_y[cur_layer];
		if (cur_layer==1)
			prev_layer_y = in;
		else
			prev_layer_y = dev[0].layer_y[prev_layer];
		cur_layer_dydx = dev[0].layer_dydx[cur_layer];
		cur_layer_dedy = dev[0].layer_dedy[cur_layer];
		prev_layer_dedy = dev[0].layer_dedy[prev_layer];
		cur_layer_dedx = dev[0].layer_dedx[cur_layer];
		cur_layer_ydedx = dev[0].layer_ydedx[cur_layer];
		cur_layer_sumdedx = dev[0].layer_sumdedx[cur_layer];
		cur_layer_bias = dev[0].bias[cur_layer];
		cur_layer_delta_bias = dev[0].delta_bias[cur_layer];
		cur_layer_delta_weights = dev[0].delta_weights[cur_layer];
		cur_weights = dev[0].weights[cur_layer];

		if (cur_layer != numlayers - 1)
		{
			//printf("former layers' sigmoid\n");
			DevDsigmoid(streams[0], cur_layer_size, cur_layer_y, cur_layer_dydx);
			DevVecMul(streams[0],   cur_layer_size, cur_layer_dydx, cur_layer_dedy, cur_layer_dedx);
		}
		//else
		//{
		
		//DevSubIndex(streams[0], n_frames, cur_layer_units, dev[0].out, targ, cur_layer_dedx);
		//��cpu���Ƶ�gpu
	  //    DevLinearOutCopy(streams[0], n_frames, cur_layer_units, dev[0].out, targ, cur_layer_dedx);
		//}
		//��ƽ������󵼣�//////////////////////////////////////////yongxu
		else
		{
		  //printf("begin to cal squared error\n");
		  //printf("targ[0]=%f,targ[1]=%f\n",targ[0],targ[1]);
			//DevSubClean(streams[0], n_frames, cur_layer_units, dev[0].layer_x[numlayers - 1], targ, cur_layer_dedx);
		  DevSubClean(streams[0], n_frames, cur_layer_units, dev[0].out, targ, cur_layer_dedx);
		  //dev[0].layer_x[numlayers - 1]
		}

		if (cur_layer != 1)
		{
			SgemmTN(handles[0], prev_layer_units, cur_layer_units, n_frames, cur_weights, cur_layer_dedx, prev_layer_dedy, zero, one);
		}

		// Update weights.
		//printf("Update weights\n");
		//SgemmNT(handles[0], cur_layer_units, n_frames, prev_layer_units, cur_layer_dedx, prev_layer_y, cur_layer_delta_weights ,momentum, -cur_lrate/n_frames);
		SgemmNT(handles[0], cur_layer_units, n_frames, prev_layer_units, cur_layer_dedx, prev_layer_y, cur_layer_ydedx ,zero, one);
		updatedelta(streams[0], cur_layer_units * prev_layer_units, cur_layer_delta_weights, cur_weights, cur_layer_ydedx, n_frames, momentum, cur_lrate, weightcost);
		//hipblasSaxpy(handles[0],cur_layer_units *prev_layer_units, &cur_lr_wc, cur_weights,1,cur_layer_delta_weights ,1);

		//DevAccSumrow(streams[0], cur_layer_units, n_frames, cur_layer_dedx, cur_layer_delta_bias, momentum, -cur_lrate/n_frames);
		DevAccSumrow(streams[0], cur_layer_units, n_frames, cur_layer_dedx, cur_layer_sumdedx, zero, one);
		updatedelta(streams[0], cur_layer_units, cur_layer_delta_bias, cur_layer_bias, cur_layer_sumdedx, n_frames, momentum, cur_lrate, zero);
		//hipblasSaxpy(handles[0],cur_layer_units, &cur_lr_wc, cur_layer_bias,1,cur_layer_delta_bias ,1);

		DevAccSum(streams[0],	cur_layer_units *prev_layer_units, cur_layer_delta_weights,	cur_weights, 1.0);		
		DevAccSum(streams[0],	cur_layer_units, cur_layer_delta_bias,	cur_layer_bias, 1.0);

		///
		/*
		if(cur_layer ==1){
		float *tmpout = new float[1 *cur_layer_units];
		fromdev_vf_vf("data",1 *cur_layer_units, cur_layer_bias,tmpout, streams[0]);
		for(int tmpj =0 ;tmpj < cur_layer_units ;tmpj ++)
		{
		for(int tmpi =0;tmpi< 1; tmpi++)
		{
		printf("%f\n",(tmpout[tmpj + tmpi *cur_layer_units]));
		}
		}
		delete [] tmpout;
		exit(0);}
		*/
		///
  //printf("come to end\n");
	}
	//fclose(fp);
}

//void BP_GPU::cv_bunch_single(int n_frames, const float *in, int* out)
void BP_GPU::cv_bunch_single(int n_frames, const float *in, float* out)///////////////////////////////by yongxu
{
    

	const float one  = 1.0f;
	//const float zero = 0.0f;
	//int i,j;
	int cur_layer;			// The index of the current layer.
	int prev_layer;			// The index of the previous layer.
	int cur_layer_units;	// The number of units in the current layer.
	int prev_layer_units;	// The number of units in the previous layer.
	int cur_layer_size;		// The size of the current layer.
  int out_dims= layersizes[numlayers-1];
  
	float* cur_layer_x;
	float* cur_layer_y;				// Output from the current layer
	const float* prev_layer_y;	// Output from the previous non-linearity.
	float* cur_layer_bias;	// Biases for the current layer.
	float* cur_weights;		// Weights inputing to the current layer.

	//int *devout;
	//devnew_vi("devout", n_frames, &devout);
	float *devout;/////////////////////////////////by yongxu
	devnew_vf("devout", n_frames*out_dims, &devout);
	
	//dropout����
	int weight_size;
	    float vis_keep;
	float hid_keep;
    vis_keep=1.0f-visible_omit;
	hid_keep=1.0f-hid_omit;
	
	//// Forward
	for (cur_layer=1; cur_layer< numlayers; cur_layer++)
	{
		prev_layer = cur_layer - 1;
		cur_layer_units = layersizes[cur_layer];
		prev_layer_units = layersizes[prev_layer];
		cur_layer_size = cur_layer_units * n_frames;
		cur_layer_x = dev[0].layer_x[cur_layer];
		cur_layer_y = dev[0].layer_y[cur_layer];
		
		 weight_size=prev_layer_units*cur_layer_units;
		
		if (cur_layer==1)
			prev_layer_y = in;
		else
			prev_layer_y = dev[0].layer_y[prev_layer];
		cur_layer_bias = dev[0].bias[cur_layer];
		
				if (dropoutflag==1)
		{
			if(cur_layer==1)
			   DevWeightMultiP(streams[0], weight_size, vis_keep, dev[0].weights[cur_layer]);
			else
				DevWeightMultiP(streams[0], weight_size, hid_keep, dev[0].weights[cur_layer]);
		}
		
		cur_weights 	 = dev[0].weights[cur_layer];

		DevMultiCopy(streams[0],n_frames, cur_layer_units, cur_layer_bias, cur_layer_x);
		SgemmNN(handles[0],cur_layer_units, prev_layer_units, n_frames, cur_weights, prev_layer_y, cur_layer_x, one, one); 
    
    		if (dropoutflag==1)
		{
            if(cur_layer==1)
			   DevWeightMultiP(streams[0], weight_size, 1.0f/vis_keep, dev[0].weights[cur_layer]);
			else 
				DevWeightMultiP(streams[0], weight_size, 1.0f/hid_keep, dev[0].weights[cur_layer]);

		}
    
		if (cur_layer != numlayers - 1){
			DevSigmoid(streams[0],cur_layer_size, cur_layer_x, cur_layer_y);
		}
		else{  /////////////////////////////////////////yongxu ע�͵��Ϳ��Եõ�һ���������������
		//	DevSoftmax(streams[0],n_frames, cur_layer_units, cur_layer_x, dev[0].out);
		//	DevGetMaxIndex(streams[0], cur_layer_units, n_frames, dev[0].out,  devout);
		//DevLinearOutCopy(streams[0],n_frames, cur_layer_units, cur_layer_x, dev[0].out);
		//hipMemcpy(dev[0].out,cur_layer_x,n_frames*cur_layer_units*sizeof(float),hipMemcpyDeviceToDevice);
		hipMemcpy(devout,cur_layer_x,n_frames*cur_layer_units*sizeof(float),hipMemcpyDeviceToDevice);
		}
	}
	//fromdev_vi_vi("devout",n_frames,devout,out, streams[0]);
	//devfree_vi("devout",devout);/////////////////////////////////////////yongxu
    fromdev_vf_vf("devout",n_frames*out_dims,devout,out, streams[0]);
	devfree_vf("devout",devout);

	////
	//		float *asf = new float[cur_layer_units* n_frames];
	//		//fromdev_vf_vf("out", cur_layer_units* n_frames, dev[0].out ,asf, streams[0]);
	//		for(int tmp=0;tmp <  n_frames;tmp++)
	//   		printf("%d\n",out[tmp]);
	//  		delete []asf;
	//   		exit(0);

}

////void BP_GPU::train_bunch_multi(int n_frames, float **in, int** targ)
//void BP_GPU::train_bunch_multi(int n_frames, float **in, float** targ)/////////////////////yongxu
//{
//	const float one  = 1.0f;
//	const float zero = 0.0f;
//	int i;
//	int cur_layer;			// The index of the current layer.
//	int prev_layer;			// The index of the previous layer.
//	
//	float cur_lrate =  lrate;
//
//	int n_frames_part[GPU_selected];
//	int part = bunchsize/GPU_selected;
//
//	for(i= 0; i< GPU_selected;i++)
//	{
//		n_frames_part[i] = part;
//	}
//	n_frames_part[GPU_selected -1] = n_frames -part*(GPU_selected -1);
//
//	for(i=0;i<GPU_selected;i++)
//	{
//		hipSetDevice(i);
//		//// Forward
//		for (cur_layer=1; cur_layer< numlayers; cur_layer++)
//		{
//	
//			prev_layer = cur_layer - 1;
//			DevMultiCopy(streams[i], n_frames_part[i], layersizes[cur_layer], dev[i].bias[cur_layer], dev[i].layer_x[cur_layer]);
//			if (cur_layer==1)       
//                SgemmNN(handles[i], layersizes[cur_layer], layersizes[prev_layer], n_frames_part[i], dev[i].weights[cur_layer], in[i], dev[i].layer_x[cur_layer], one, one); 
//			else
//			    SgemmNN(handles[i], layersizes[cur_layer], layersizes[prev_layer], n_frames_part[i], dev[i].weights[cur_layer], dev[i].layer_y[prev_layer], dev[i].layer_x[cur_layer], one, one); 
//			
//			if (cur_layer != numlayers - 1){
//				DevSigmoid(streams[i], layersizes[cur_layer] * n_frames_part[i], dev[i].layer_x[cur_layer], dev[i].layer_y[cur_layer]);
//			}
//			//else{ /////////////////////////////////yongxu, ע�͵����ܵõ����������
//			//	DevSoftmax(streams[i],n_frames_part[i], layersizes[cur_layer], dev[i].layer_x[cur_layer], dev[i].out);
//			//}
//			
//
//		}
//
//		// Backward
//		for (cur_layer = numlayers -1; cur_layer >0; cur_layer--)
//		{
//			prev_layer = cur_layer - 1;
//
//
//			if (cur_layer != numlayers - 1)
//			{
//				DevDsigmoid(streams[i], layersizes[cur_layer] * n_frames_part[i], dev[i].layer_y[cur_layer], dev[i].layer_dydx[cur_layer]);
//				DevVecMul(streams[i],   layersizes[cur_layer] * n_frames_part[i], dev[i].layer_dydx[cur_layer], dev[i].layer_dedy[cur_layer], dev[i].layer_dedx[cur_layer]);
//
//			}
//			//else/////////////////////////////////yongxu, ע�͵����ܵõ����������
//			//{
//			//
//			//	DevSubIndex(streams[i], n_frames_part[i], layersizes[cur_layer], dev[i].out, targ[i], dev[i].layer_dedx[cur_layer]);
//			//	
//			//}
//		    //��ƽ������󵼣�//////////////////////////////////////////yongxu
//		    else
//		    {
//		
//		    	DevSubClean(streams[i], n_frames_part[i], layersizes[cur_layer], dev[i].layer_x[numlayers - 1], targ[i], dev[i].layer_dedx[cur_layer]);
//		
//		     }
//
//			if (cur_layer != 1)
//			{
//				SgemmTN(handles[i], layersizes[prev_layer], layersizes[cur_layer], n_frames_part[i], dev[i].weights[cur_layer], dev[i].layer_dedx[cur_layer], dev[i].layer_dedy[prev_layer], zero, one);
//				
//			}
//
//			// Update weights.
//			if (cur_layer ==1)
//		        SgemmNT(handles[i], layersizes[cur_layer], n_frames_part[i], layersizes[prev_layer], dev[i].layer_dedx[cur_layer], in[i], dev[i].layer_ydedx[cur_layer] ,zero, one);
//			else
//			    SgemmNT(handles[i], layersizes[cur_layer], n_frames_part[i], layersizes[prev_layer], dev[i].layer_dedx[cur_layer], dev[i].layer_y[prev_layer], dev[i].layer_ydedx[cur_layer] ,zero, one);
//		    DevAccSumrow(streams[i], layersizes[cur_layer], n_frames_part[i], dev[i].layer_dedx[cur_layer], dev[i].layer_sumdedx[cur_layer], zero, one);
//
//		}
//	}
//	hipDeviceSynchronize();
//	hipSetDevice(0);
//	
//	for(i= 1; i< GPU_selected;i++)
//	{
//		hipDeviceEnablePeerAccess(i, 0);
//		for (cur_layer=1; cur_layer< numlayers; cur_layer++)
//		{
//		    prev_layer = cur_layer - 1;
//
//			hipblasSaxpy(handles[0],layersizes[cur_layer] * layersizes[prev_layer], &one, dev[i].layer_ydedx[cur_layer], 1, dev[0].layer_ydedx[cur_layer] , 1);
//			hipblasSaxpy(handles[0],layersizes[cur_layer], &one, dev[i].layer_sumdedx[cur_layer], 1, dev[0].layer_sumdedx[cur_layer] , 1);
//
//		} 
//	}
//	hipDeviceSynchronize();
//	for (cur_layer=1; cur_layer< numlayers; cur_layer++)
//	{
//		prev_layer = cur_layer - 1;
//
//		updatedelta(streams[0], layersizes[cur_layer] * layersizes[prev_layer], dev[0].delta_weights[cur_layer], dev[0].weights[cur_layer], dev[0].layer_ydedx[cur_layer], n_frames, momentum, cur_lrate, weightcost);
//		updatedelta(streams[0], layersizes[cur_layer], dev[0].delta_bias[cur_layer], dev[0].bias[cur_layer], dev[0].layer_sumdedx[cur_layer], n_frames, momentum, cur_lrate, zero);
//		DevAccSum(streams[0],	layersizes[cur_layer] * layersizes[prev_layer], dev[0].delta_weights[cur_layer],	dev[0].weights[cur_layer], 1.0);		
//		DevAccSum(streams[0],	layersizes[cur_layer], dev[0].delta_bias[cur_layer],	dev[0].bias[cur_layer], 1.0);
//	}
//	//hipStreamSynchronize(streams[0]);
//
//	////copy paras to other gpus
//	for(i= 1; i< GPU_selected;i++)
//	{
//	    //hipSetDevice(i);
//		//hipDeviceEnablePeerAccess(i, 0);
//		for (cur_layer=1; cur_layer< numlayers; cur_layer++)
//		{
//			prev_layer = cur_layer - 1;
//
//			hipblasScopy(handles[0], layersizes[cur_layer] * layersizes[prev_layer], dev[0].weights[cur_layer],1,dev[i].weights[cur_layer] ,1);
//			hipblasScopy(handles[0], layersizes[cur_layer], dev[0].bias[cur_layer],1, dev[i].bias[cur_layer] ,1);
//
//
//			hipblasScopy(handles[0],layersizes[cur_layer] * layersizes[prev_layer], dev[0].delta_weights[cur_layer],1,dev[i].delta_weights[cur_layer] ,1);
//			hipblasScopy(handles[0],layersizes[cur_layer], dev[0].delta_bias[cur_layer],1, dev[i].delta_bias[cur_layer] ,1);
//		}			
//
//	}
//	hipStreamSynchronize(streams[0]);
//	hipDeviceSynchronize();
//
//}

void BP_GPU::returnWeights(float **weights, float **bias)
{
	int i;
	////copy weights && biases to devices

	hipSetDevice(0);
  //hipSetDevice(1);
   
	for(i = 1; i< numlayers; i++)
	{
		fromdev_vf_vf("weights", layersizes[i-1] *layersizes[i], dev[0].weights[i], weights[i], streams[0]);
		fromdev_vf_vf("bias", layersizes[i], dev[0].bias[i], bias[i], streams[0]);
	}
}

///// following are alloc and free functions
void BP_GPU::devnew_vf(const char* varname, int n, float **devptr)
{
	hipError_t cudaStat =  hipMalloc((void **) devptr, n* sizeof(float));
	if(cudaStat !=hipSuccess ) 
	{
		printf("%s device momory alloc error\n", varname);
		exit(0);
	}
	//float *zero = new float [n];
	float *zero;
	hipHostMalloc((void**)&zero,n*sizeof(float));

	for(int i=0;i< n;i++)
		zero[i] = 0.0f;
	hipblasSetVector(n,sizeof(float),zero,1,(*devptr),1);
	//delete []zero;
	hipHostFree(zero);
}

void BP_GPU::devnew_vi(const char* varname, int n, int **devptr)
{
	hipError_t cudaStat = hipMalloc((void **) devptr, n* sizeof(int));
	if(cudaStat !=hipSuccess ) 
	{
		printf( "%s device momory alloc error\n", varname);
		exit(0);
	}
	//int *zero = new int [n];
	int *zero;
	hipHostMalloc((void**)&zero,n*sizeof(int));

	for(int i=0;i< n;i++)
		zero[i] = 0;
	hipblasSetVector(n,sizeof(int),zero,1,(*devptr),1);
	//delete []zero;
	hipHostFree(zero);
}

void BP_GPU::devfree_vf(const char* varname, float* devptr)
{
	hipFree((void *) devptr);
}

void BP_GPU::devfree_vi(const char* varname, int* devptr)
{
	hipFree((void *) devptr);
}

void BP_GPU::todev_vf_vf(const char* varname, int n, const float* from, float* devto, hipStream_t stream)
{
	hipblasStatus_t  e = hipblasSetVectorAsync(n, sizeof(float), from, 1, devto, 1, stream);
	if (e != HIPBLAS_STATUS_SUCCESS)
	{
		printf("cuda blas todev_vf_vf error variable %s\n",varname);
		exit(0);
	}
}

void BP_GPU::fromdev_vf_vf(const char* varname, int n, const float* devfrom, float* to, hipStream_t stream)
{
	hipblasStatus_t e = hipblasGetVectorAsync(n, sizeof(float), devfrom, 1, to, 1, stream);
	if (e != HIPBLAS_STATUS_SUCCESS)
	{
		printf("cuda blas fromdev_vf_vf error variable %s\n",varname);
		exit(0);
	}
}

//void BP_GPU::todev_vi_vi(const char* varname, int n, const int* from,int *devto, hipStream_t stream)
//{
//	hipblasStatus_t e = hipblasSetVectorAsync(n, sizeof(int), from, 1, devto, 1, stream);
//	if (e != HIPBLAS_STATUS_SUCCESS)
//	{
//		printf("cuda blas todev_vi_vi error variable %s\n", varname);
//		exit(0);
//	}
//}

//void BP_GPU::fromdev_vi_vi(const char* varname, int n,const int* devfrom, int* to, hipStream_t stream)
//{
//	hipblasStatus_t e = hipblasGetVectorAsync(n, sizeof(int), devfrom, 1, to, 1, stream);
//	if (e != HIPBLAS_STATUS_SUCCESS)
//	{
//		printf("cuda blas fromdev_vi_vi error variable %s\n", varname);
//		exit(0);
//	}
//}
